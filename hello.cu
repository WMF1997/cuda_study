#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloFromGPU(void) {
	printf("hello from GPU\n");
	return;	
}

void HelloFromCPU(void) {
	printf("hello from CPU\n");
	return;	
}

int main (void) {
	HelloFromCPU();
	HelloFromGPU<<<2, 5>>>();
	HelloFromCPU();
	return 0;
}
